#include "hip/hip_runtime.h"
#include <iostream>
#include<stdio.h>

#include<hip/hip_runtime.h>
#include<Windows.h>
#include<hip/hip_runtime.h>
#include <>
#include<glm/glm.hpp>

#include<vector>
using namespace std;
#define BLOCK 512


//�΂˒萔
#define K 1.2
//������
#define lambda 0.99
//���x�͂������ɂ�����

float (*d_v)[3];
float (*h_v)[3];

float(*d_realpoints)[3];
float(*h_realpoints)[3];
float(*h_MV)[4];
float(*d_MV)[4];

float *h_dbg;
float *d_dbg;


__global__ void d_spring(unsigned int num_points, float(*realpoints)[3], float(*virtualpoints)[3], float(*vel)[3], float dt, float(*M)[4], float* dbg);


void initField(unsigned int num_points,vector<float>& position) {

	h_v =(float(*)[3])malloc(num_points *3 * sizeof(float));
	h_realpoints =(float(*)[3])malloc(num_points *3 * sizeof(float));
	h_MV =(float(*)[4])malloc(4*4 * sizeof(float));
	h_dbg = (float*)malloc(sizeof(float));
	
	hipMalloc((void**)&d_v, sizeof(float)*3 * num_points);
	hipMalloc((void**)&d_realpoints, sizeof(float)*3 * num_points);
	hipMalloc((void**)&d_MV, sizeof(float)*4*4);

	hipMalloc((void**)&d_dbg, sizeof(float));

	for (int i = 0; i < num_points; i++) {
		for (int j = 0; j < 3; j++) {
			h_v[i][j] = 0.0f;
			h_realpoints[i][j] = position[i * 3 + j];
			//h_realpoints[i][j] = 2.0f;
		}
	}

	hipMemcpy(d_v, h_v, num_points * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_realpoints, h_realpoints, num_points * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_MV, h_MV, 4*4 * sizeof(float), hipMemcpyHostToDevice);
}

void launchVertexProcess(unsigned int NUM_POINTS,float (*virtualpoints)[3],float dt,glm::mat4& M) {
	dim3 grid(NUM_POINTS /512 + 1, 1);
	dim3 block(512, 1, 1);

	

	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			h_MV[i][j]=M[j][i];

		}
		
	}
	hipMemcpy(d_MV, h_MV, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);



	d_spring <<< grid, block >>> (NUM_POINTS,d_realpoints,virtualpoints,d_v,dt, d_MV,d_dbg);

	
	//hipMemcpy(h_dbg, d_dbg, sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(h_v, d_v, NUM_POINTS * 3 * sizeof(float), hipMemcpyDeviceToHost);
	//printf("[%.10f]\n", h_dbg[0]);
	//std::cout << h_realpoints[0][0] << std::endl;

	//printf("finish\n");
}

__global__ void d_spring(unsigned int num_points, float(*realpoints)[3], float(*virtualpoints)[3], float(*vel)[3], float dt, float(*M)[4], float* dbg) {
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;


	if (index > num_points) {
		return;
	}

	float p[3];

	p[0] = M[0][0] * realpoints[index][0] + M[0][1] * realpoints[index][1] + M[0][2] * realpoints[index][2] + M[0][3] * 1.0f;
	p[1] = M[1][0] * realpoints[index][0] + M[1][1] * realpoints[index][1] + M[1][2] * realpoints[index][2] + M[1][3] * 1.0f;
	p[2] = M[2][0] * realpoints[index][0] + M[2][1] * realpoints[index][1] + M[2][2] * realpoints[index][2] + M[2][3] * 1.0f;
	float w = M[3][0] * realpoints[index][0] + M[3][1] * realpoints[index][1] + M[3][2] * realpoints[index][2] + M[3][3] * 1.0f;


	for (int i = 0; i < 3; i++) {
		//p[i] /= w;
		vel[index][i] += -K * dt * (virtualpoints[index][i] - p[i]);
		virtualpoints[index][i] += vel[index][i] * dt;
		vel[index][i] *= lambda;

	}
	/*
	if (index == 0) {
		dbg[0] = 2.0f;
	}
	*/

}
