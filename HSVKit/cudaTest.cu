#include "hip/hip_runtime.h"

#include <iostream>
#include<stdio.h>

#include<hip/hip_runtime.h>
#include<Windows.h>
#include<hip/hip_runtime.h>
#include <>
#define BLOCK 16
#define WIDTH 1024

float h_A[WIDTH * WIDTH];
float h_B[WIDTH * WIDTH];
float h_C[WIDTH * WIDTH];

float* d_A, * d_B, * d_C;



void h_multiply(float* A, float* B, float* C);
__global__ void d_multiply0(float* A, float* B, float* C);




void cudaTest()
{
	unsigned int i;

	hipMalloc((void**)&d_A, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_B, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_C, sizeof(float) * WIDTH * WIDTH);
	for (i = 0; i < (WIDTH * WIDTH); i++) {
		h_A[i] = (float)i;
		h_B[i] = (float)i;
	}

	LARGE_INTEGER freq;
	QueryPerformanceFrequency(&freq);

	LARGE_INTEGER start, end;

	QueryPerformanceCounter(&start);

	hipMemcpy(d_A, h_A, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(d_A, h_A, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);

	dim3 grid(WIDTH / BLOCK, WIDTH / BLOCK);
	dim3 block(BLOCK, BLOCK);

	d_multiply0 <<< grid, block >>> (d_A, d_B, d_C);

	hipMemcpy(h_C, d_C, sizeof(float) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

	QueryPerformanceCounter(&end);

	double time = static_cast<double>(end.QuadPart - start.QuadPart) * 1000.0 / freq.QuadPart;
	printf("�f�o�C�X�v������ %lf[ms]\n", time);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	QueryPerformanceCounter(&start);
	h_multiply(h_A, h_B, h_C);
	QueryPerformanceCounter(&end);

	time = static_cast<double>(end.QuadPart - start.QuadPart) * 1000.0 / freq.QuadPart;
	printf("�z�X�g�v������ %lf[ms]\n", time);
	printf("�z�X�g�v�Z����: %f\n", h_C[WIDTH * WIDTH - 1]);

}

__global__ void d_multiply0(float* A, float* B, float* C) {
	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int i;
	float tmp = 0.0;
	for (i = 0; i < WIDTH; i++) {
		tmp += A[WIDTH * r + i] * B[WIDTH * i + c];

	}
	C[WIDTH * r + c] = tmp;
}

void h_multiply(float* A, float* B, float* C) {
	unsigned int r, c, i;
	float tmp;
	for (r = 0; r < WIDTH; r++) {
		for (c = 0; c < WIDTH; c++) {
			tmp = 0.0;
			for (i = 0; i < WIDTH; i++) {
				tmp += A[WIDTH * r + i] * B[WIDTH * i + c];

			}
			C[WIDTH * r + c] = tmp;
		}
	}
}